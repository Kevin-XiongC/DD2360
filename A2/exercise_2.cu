
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <sys/time.h>
#define N 1<<17
#define margin 1e-6



double cpuSecond() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void SAXPY (float* x,float* y,float* a){
//AX+Y
    int id =blockIdx.x*blockDim.x+threadIdx.x;
    if(id<N)
        y[id]+=*a+x[id];
}

int main(){   
    float* x,*y,a,*r;
    uint gridsize=(N+256-1)/256;
    uint nBytes=sizeof(float)*N;
    x=(float*)malloc(nBytes);
    y=(float*)malloc(nBytes);
    r=(float*)malloc(nBytes);
    for(int i=0;i<N;i++)
    {
        x[i]=i;
        y[i]=2*i;
    }
    a=1.1;

    float *d_x, *d_y, *d_a;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_a, sizeof(float));
    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_a, (void*)&a, sizeof(float), hipMemcpyHostToDevice);

    double start = cpuSecond();
    for(int i=0;i<N;i++)
    {
        y[i]+=a*x[i];
        r[i]=y[i];
    }
    printf("CPU costs %lf\n",cpuSecond()-start);

    start=cpuSecond();
    SAXPY<<<gridsize,256>>>(d_x, d_y, d_a);
    hipDeviceSynchronize();
    printf("GPU costs %lf\n",cpuSecond()-start);
    hipMemcpy((void*)y, (void*)d_y, nBytes, hipMemcpyDeviceToHost);

    int c=0;
    for(int i=0;i<N;i++)
        c+=abs(y[i]-r[i])>margin?1:0;
    printf("%d mismatches\n");
      
    hipFree(d_a);
    hipFree(d_x);
    hipFree(d_y);
    return 0;
}
